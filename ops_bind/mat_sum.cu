#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "mat_sum.h"

template<class T> 
__global__ void vecaddOnDevice(T *a, T *b, T *c, const int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}


__host__ void gpuMatAdd(char* lv, char* rv, char* res, size_t dataSize)
{
	dim3 block(512);
	dim3 grid((dataSize + block.x - 1) / block.x);
	vecaddOnDevice<char> <<<grid, block >>> (lv, rv, res, dataSize);
}